#include "hip/hip_runtime.h"
#ifndef RGBTOYV12_CU_
#define RGBTOYV12_CU_

#include "RGBtoYV12.h"


// Dieser Kernel wird mit
// RGBtoYV12<<<gridSize, blockSize>>>(yuv, devPtr);
//aufgerufen.

//er berechnet direkt die YUV-Werte aus dem RGBA- Format
// (Hoffentlich nun direkt richtigrum und nicht mehr spiegelverkehrt.

__global__ void RGBtoYV12(unsigned char* yuv, unsigned char* pData)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
    int width = gridDim.x * blockDim.x * 1.5;
	//bildbreite
	int iwidth = gridDim.x;

 	int rgbID = i * 4;
    int upos = blockDim.x * gridDim.x;
    int vpos = upos + upos / 4;
	int col = i % iwidth;
	int row = i / iwidth; //bzw. threadIdx.x;

 	int r = pData[rgbID], g = pData[rgbID+1], b = pData[rgbID+2];


	//Y
	// neu (hoffentlich richtig rum und nicht gespiegelt)
	// nur noch zeilenweise umgedreht. die eigene ID steckt in
	// den Zeilen und Spalten, und taucht deshalb nicht extra auf.

	int y =  ((  66 * r + 129 * g +  25 * b + 128) >> 8) +  16;
	yuv[upos - (row+1)*iwidth + col] = y;


     if (           !((i/gridDim.x)%2)             &&        !(i%2))
     {

        //YV12
 	    // U
		int u = ((-38 * r - 74 * g + 112 * b + 128) >> 8) + 128;
		yuv[width - ( (iwidth/2) * ((row/2)+1) - ((col/2)+1)  )] = u;
        // V
		int v = ((112 * r - 94 * g - 18 * b + 128) >> 8) + 128;
		yuv[vpos - ( (iwidth/2) * ((row/2)+1) - ((col/2)+1)  )] = v; 

    }
}


extern void callKernel(int width, int height, unsigned char* yuv, unsigned char* devPtr)
{
	RGBtoYV12<<<width, height>>>(yuv, devPtr);
    hipDeviceSynchronize();
}

#endif

